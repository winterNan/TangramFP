#include "hip/hip_runtime.h"
// kacy16_conv2d_cuda.cu --- 
// 
// Filename: kacy16_conv2d_cuda.cu
// Description: 
// Author: Yuan Yao <yuan.yao@it.uu.se>
// Maintainer: 
// Created: Thu Apr 25 14:35:19 2024 (+0200)
// Version: 
// Package-Requires: ()
// Last-Updated: 
//           By: 
//     Update #: 0

// Change Log:
// 
// 
// 
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or (at
// your option) any later version.
// 
// This program is distributed in the hope that it will be useful, but
// WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
// General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with GNU Emacs.  If not, see <https://www.gnu.org/licenses/>.
// 
// 

// Code:

#include <torch/types.h>
#include "pytorch_cuda_helper.h"

template <typename T>
__global__ void kacy16_conv_im2col_gpu_kernel(
    const int n, const T *data_im, const int height,
    const int width, const int kernel_h, const int kernel_w, const int pad_h,
    const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int batch_size,
    const int num_channels, const int height_col,
    const int width_col, T *data_col)
{
    CUDA_1D_KERNEL_LOOP(index, n)
    {
        // index index of output matrix
        const int w_col = index % width_col;
        const int h_col = (index / width_col) % height_col;
        const int b_col = (index / width_col / height_col) % batch_size;
        const int c_im = (index / width_col / height_col) / batch_size;
        const int c_col = c_im * kernel_h * kernel_w;

        const int h_in = h_col * stride_h - pad_h;
        const int w_in = w_col * stride_w - pad_w;
        T *data_col_ptr =
            data_col +
            ((c_col * batch_size + b_col) * height_col + h_col) * width_col + w_col;
        const T *data_im_ptr =
            data_im + (b_col * num_channels + c_im) * height * width;

        for (int i = 0; i < kernel_h; ++i)
        {
            for (int j = 0; j < kernel_w; ++j)
            {
                T val = static_cast<T>(0);
                const int h_im = h_in + i * dilation_h;
                const int w_im = w_in + j * dilation_w;
                if (h_im > -1 && w_im > -1 && h_im < height && w_im < width)
                {
                    val = data_im_ptr[h_im * width + w_im];
                }
                *data_col_ptr = val;
                data_col_ptr += batch_size * height_col * width_col;
            }
        }
    }
}

void kacy16_conv_im2col_cuda(Tensor data_im,
                         const int channels, const int height,
                         const int width, const int ksize_h,
                         const int ksize_w, const int pad_h, const int pad_w,
                         const int stride_h, const int stride_w,
                         const int dilation_h, const int dilation_w,
                         const int parallel_imgs, Tensor data_col)
{
    int height_col =
        (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
    int width_col =
        (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
    int num_kernels = channels * height_col * width_col * parallel_imgs;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        data_im.scalar_type(), "kacy16_conv_im2col_gpu", [&]
        { kacy16_conv_im2col_gpu_kernel<scalar_t><<<GET_BLOCKS(num_kernels),
                                                THREADS_PER_BLOCK, 0,
                                                at::cuda::getCurrentCUDAStream()>>>(
              num_kernels, data_im.data_ptr<scalar_t>(),
              height, width, ksize_h, ksize_w,
              pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
              parallel_imgs, channels,
              height_col, width_col, data_col.data_ptr<scalar_t>()); });

    AT_CUDA_CHECK(hipGetLastError());
}

// 
// kacy16_conv2d_cuda.cu ends here
